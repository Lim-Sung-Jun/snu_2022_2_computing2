#include "hip/hip_runtime.h"
#include "namegen.h"
#include "util.h"

#include <cassert>
#include <math.h>
#include <vector>

#include <mpi.h>

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }

#define MAX_NUM_GPU 4
#define TILE_WIDTH 16
#define BATCHSIZE 16
int num_devices = 0;

// Array of device (GPU) pointers
static float *a_d[MAX_NUM_GPU];
static float *b_d[MAX_NUM_GPU];
static float *c_d[MAX_NUM_GPU];
static int Mbegin[MAX_NUM_GPU], Mend[MAX_NUM_GPU];

// Defined in main.cpp
extern int mpi_rank, mpi_size;

// You can modify the data structure as you want
struct Tensor {

  // constructor
  /* Alloc memory */
  Tensor(std::vector<int> shape_) {
    ndim = shape_.size();
    for (size_t i = 0; i < ndim; i++) {
      shape[i] = shape_[i];
    }

    size_t n = num_elem();
    buf = (float *)malloc(n * sizeof(float)); //할당한 shape만큼 메모리에 공간을 준비한다.
  }

  /* Alloc memory and copy */
  Tensor(std::vector<int> shape_, float *buf_) {
    ndim = shape_.size();
    for (size_t i = 0; i < ndim; i++) {
      shape[i] = shape_[i];
    }

    size_t n = num_elem();
    buf = (float *)malloc(n * sizeof(float));
    memcpy(buf, buf_, n * sizeof(float)); // 해당 공간을 buf로 옮긴다.
  }

  // destructor
  ~Tensor() {
    if (buf != nullptr)
      free(buf);
  }

  void set_SOS() {
    size_t n = num_elem();
    for (size_t i = 0; i < n; i++)
      buf[i] = SOS; // 데이터를 모두 0으로 바꾼다.
  }

  void set_zero() {
    size_t n = num_elem();
    for (size_t i = 0; i < n; i++)
      buf[i] = 0.0; // 데이터를 모두 0으로 바꾼다.
  }

  size_t num_elem() {
    size_t sz = 1;
    for (size_t i = 0; i < ndim; i++)
      sz *= shape[i];
    return sz; // 전체 갯수를 센다.
  }

  // Pointer to data
  float *buf = nullptr; // 데이터를 가리킨다.

  // Shape of tensor, from outermost dimension to innermost dimension.
  // e.g., {{1.0, -0.5, 2.3}, {4.3, 5.6, -7.8}} => shape = {2, 3}
  size_t ndim = 0;
  size_t shape[4];
};

/* Network parameters */
Tensor *character_embedding;
Tensor *W_ir0, *W_iz0, *W_in0, *W_ir1, *W_iz1, *W_in1;
Tensor *W_hr0, *W_hz0, *W_hn0, *W_hr1, *W_hz1, *W_hn1;
Tensor *b_ir0, *b_iz0, *b_in0, *b_ir1, *b_iz1, *b_in1;
Tensor *b_hr0, *b_hz0, *b_hn0, *b_hr1, *b_hz1, *b_hn1;

Tensor *b_ir0_stack, *b_iz0_stack, *b_in0_stack, *b_ir1_stack, *b_iz1_stack, *b_in1_stack;
Tensor *b_hr0_stack, *b_hz0_stack, *b_hn0_stack, *b_hr1_stack, *b_hz1_stack, *b_hn1_stack, *b_fc_stack;
Tensor *W_fc, *b_fc;
Tensor *rfloats;

/* input, activations, output */
Tensor *input, *emb_out;
Tensor *hidden0, *hidden1;
Tensor *r0, *r1, *z0, *z1, *n0, *n1, *f, *char_prob;
Tensor *rtmp00, *rtmp01, *rtmp02, *rtmp03, *rtmp04;
Tensor *rtmp10, *rtmp11, *rtmp12, *rtmp13, *rtmp14;
Tensor *ztmp00, *ztmp01, *ztmp02, *ztmp03, *ztmp04;
Tensor *ztmp10, *ztmp11, *ztmp12, *ztmp13, *ztmp14;
Tensor *ntmp00, *ntmp01, *ntmp02, *ntmp03, *ntmp04, *ntmp05;
Tensor *ntmp10, *ntmp11, *ntmp12, *ntmp13, *ntmp14, *ntmp15;
Tensor *htmp00, *htmp01, *htmp02;
Tensor *htmp10, *htmp11, *htmp12;
Tensor *ftmp0;

/* Operations */

/*
 * Embedding
 * input: [1] (scalar)
 * weight: [NUM_CHAR x EMBEDDING_DIM]
 * output: [EMBEDDING_DIM]
 */
void embedding(Tensor *input, Tensor *weight, Tensor *output) {
  // size_t n = weight->shape[1]; // weight에 shape도 있고 buf가 있다.
  // for (size_t i = 0; i < n; i++) {
  //   int x = (int)input->buf[0];  // 현재 문자가 무엇인지 알아낸다.
  //   output->buf[i] = weight->buf[x * n + i];
  // }
  size_t input_N = input->shape[1];
  size_t n = weight->shape[1]; // weight에 shape도 있고 buf가 있다.
  for(size_t i = 0; i < input_N; i++){
    for (size_t j = 0; j < n; j++) {
        int x = (int)input->buf[i];  // 현재 문자가 무엇인지 알아낸다.
        output->buf[j * input_N + i] = weight->buf[x * n + j];
        // printf("output->buf[%zd * %zd + %zd]: %f\n",j,input_N,i,output->buf[j * input_N + i]);
    }
  }  
}
/*
 * Elementwise addition
 * input1: [*]
 * input2: [*] (same shape as input1)
 * output: [*] (same shape as input1)
 */
void elemwise_add(Tensor *input1, Tensor *input2, Tensor *output) {
  size_t sn = input1->num_elem();
  for (size_t i = 0; i < sn; i++) {
    output->buf[i] = input1->buf[i] + input2->buf[i];
  }
}

/*
 * Elementwise (1-x)
 * input: [*]
 * output: [*] (same shape as input)
 */
void elemwise_oneminus(Tensor *input, Tensor *output) {
  size_t n = input->num_elem();
  for (size_t i = 0; i < n; i++) {
    float x = input->buf[i];
    output->buf[i] = 1.0 - x;
  }
}

/*
 * Elementwise multiplication
 * input1: [*]
 * input2: [*] (same shape as input1)
 * output: [*] (same shape as input1)
 */
void elemwise_mul(Tensor *input1, Tensor *input2, Tensor *output) {
  size_t sn = input1->num_elem();
  for (size_t i = 0; i < sn; i++) {
    output->buf[i] = input1->buf[i] * input2->buf[i];
  }
}

/*
 * Elementwise tanh(x)
 * input: [*]
 * output: [*] (same shape as input)
 */
void elemwise_tanh(Tensor *input, Tensor *output) {
  size_t n = input->num_elem();
  for (size_t i = 0; i < n; i++) {
    float x = input->buf[i];
    output->buf[i] = tanhf(x);
  }
}

/*
 * Elementwise Sigmoid 1 / (1 + exp(-x))
 * input: [*]
 * output: [*] (same shape as input)
 */
void elemwise_sigmoid(Tensor *input, Tensor *output) {
  size_t n = input->num_elem();
  for (size_t i = 0; i < n; i++) {
    float x = input->buf[i];
    output->buf[i] = 1.0 / (1.0 + expf(-x));
  }
}

/*
 * SGEMV
 * input1: [N x K]
 * input2: [K]
 * output: [N]
 */
void matvec(Tensor *input1, Tensor *input2, Tensor *output) {
  size_t N_ = input1->shape[0];
  size_t K_ = input1->shape[1];
  for (size_t i = 0; i < N_; i++) {
    float c = 0.0;
    for (size_t j = 0; j < K_; j++) {
      c += input1->buf[i * K_ + j] * input2->buf[j];
    }
    output->buf[i] = c;
  }
}

/*
 * SGEMM
 * input1: [M x K]
 * input2: [K x N]
 * output: [M x N]
 */
void matmul(Tensor *input1, Tensor *input2, Tensor *output) { // tiled 사용하기
  size_t M_ = input1->shape[0];
  size_t K_ = input1->shape[1];
  size_t N_ = input2->shape[1];

  // int M_start = (N_ / mpi_size) * mpi_rank;
  // int M_end = (N_ / mpi_size) * (mpi_rank + 1);
  // printf("mpi_rank[%d]: start:%d, end:%d\n",mpi_rank, M_start, M_end);
  // // node마다 행렬을 나눠줬다.
  // if(mpi_rank == 0){
  //   #pragma omp parallel for num_threads(mpi_size - 1)
  //   for(int i = 1; i < mpi_size; i++)
  //   {
  //     // 1,2,3 rank의 start, end값은 모르기때문에 이렇게 처리해준다.
  //     int M_start = i * (N_ / mpi_size);
  //     int M_end = (i + 1) * (N_ / mpi_size);
  //     MPI_Send(input1->buf + M_start * K_, (M_end - M_start) * K_, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
  //     MPI_Send(input2->buf, K_*N_, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
  //   }

  // }else{
  //   MPI_Recv((void*)(input1->buf + M_start * K_), (M_end - M_start) * K_, MPI_FLOAT, 0, 1, MPI_COMM_WORLD,MPI_STATUS_IGNORE);
  //   MPI_Recv((void*)input2->buf, K_*N_, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
  // }
  #pragma omp parallel for
  for (size_t i = 0; i < M_; i++) {
    for (size_t j = 0; j < N_; j++) {
      float c = 0.0;
      for (size_t k = 0; k < K_; k++) {
        c += input1->buf[i * K_ + k] * input2->buf[k * N_ + j];
      }
      output->buf[i * N_ + j] = c;
    }
  }

  // int size = N_ / mpi_size;
  // if(mpi_rank == 0){
  //   #pragma omp parallel for num_threads(mpi_size - 1)
  //   for(int i = 1; i < mpi_size; i++)
  //   {
  //     int M_start = i * (N_ / mpi_size);
  //     // int M_end = (i + 1) * (M / mpi_world_size);
  //     // MPI_Send(A + M_start * K, (M_end - M_start) * K, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
  //     // MPI_Send(B, K*N, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
  //     MPI_Recv(output->buf + M_start * N_, size * N_, MPI_FLOAT, i, i, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
  //     printf("[mpi_rank:%d] recv address:%zd\n",mpi_rank, M_start * N_);
  //   }
  // }else{
  //   MPI_Send(output->buf + M_start * N_, size * N_, MPI_FLOAT, 0, mpi_rank, MPI_COMM_WORLD);
  //   printf("[mpi_rank:%d] send address:%zd\n",mpi_rank, M_start * N_);
  // }
}

__global__ void matmul_kernel(float *A, float *B, float *C, const int M, const int K, const int N)
{
    __shared__ float Asub[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bsub[TILE_WIDTH][TILE_WIDTH];
 
    int by = blockIdx.x, bx = blockIdx.y;
    int ty = threadIdx.x, tx = threadIdx.y;
 
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
 
    float Pvalue = 0;
    for (int ph = 0; ph < ceil(K / (float)TILE_WIDTH); ++ph) {
        Asub[ty][tx] = A[Row*K + ph*TILE_WIDTH + tx];
        Bsub[ty][tx] = B[(ph*TILE_WIDTH + ty)*K + Col];
 
        __syncthreads();
 
        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += Asub[ty][k] * Bsub[k][tx];
        }
 
        __syncthreads();
    }
    C[Row*K + Col] = Pvalue;
}

/*
 * Softmax
 * Normalize the input elements according to its exp value.
 * The result can be interpreted as a probability distribution.
 * input: [*]
 * output: [*], (same shape as input)
 */
void softmax(Tensor *input, Tensor *output, int N) { // softmax(f, char_prob, BATCHSIZE);
  for(int k = 0; k < N; k++){
    size_t n = input->num_elem()/N;
    float sum = 0.0;
    for (size_t i = 0; i < n; i++) {
      float x = input->buf[i*N+k];
      sum += expf(x);
    }
    for (size_t i = 0; i < n; i++) {
      float x = input->buf[i*N+k];
      output->buf[i*N+k] = expf(x) / sum;
    }
  }
}

/*
 * Sample a random index according to the given probability distribution 
 * This function is called at most N*MAX_LEN times. Each call uses a
 * random float in [0,1] to sample an index from the given distribution.
 * input: [NUM_CHAR], probability distribution of the characters
 * rng_seq: [N*MAX_LEN],
 */
void random_select(Tensor *char_prob, Tensor *rng_seq, Tensor *input, char *output, int N, int l) { //char_prob, rfloats, n * MAX_LEN + l // 글씨의 최대 길이. ??
//char_prob, rfloats, input, output, N, l);n * MAX_LEN + l
  int selected_char = 0;
  for(int n = 0; n < BATCHSIZE; n++){ // 64 // 여기 batchsize는 전체 양을 node로 나눈 수
    int rng_offset = (n + (mpi_rank * BATCHSIZE)) * MAX_LEN + l; // 여기에 mpi를 고려한다.
    float r = rng_seq->buf[rng_offset]; //
    size_t n_elem = char_prob->num_elem()/BATCHSIZE;
    float psum = 0.0;
    for (size_t i = 0; i < n_elem; i++) {
      
      psum += char_prob->buf[i * BATCHSIZE + n];
      
      if (psum > r) {
        selected_char = i;
        break;
      }
    }
    if(psum <= r){
      selected_char = n_elem - 1;
    }
    // printf("psum:%f, r: %f, selected_char: %d\n",psum, r,selected_char);
    // printf("mpirank:%d output index: %d\n",mpi_rank,(n + (mpi_rank * BATCHSIZE)));
    output[(n + (mpi_rank * BATCHSIZE)) * (MAX_LEN + 1) + l] = selected_char; //
    input->buf[n] = selected_char; // 다시 입력으로 넣는다.
  }
}

void stack_vector(int N, Tensor *b_ir0_stack){
  // Tensor* b_ir0_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
  int size = HIDDEN_DIM;
  if(b_ir0_stack->shape[0] == NUM_CHAR){
    size = NUM_CHAR;
  }
  for(int i = 0; i < N; i++){
    for(int j = 0; j < size; j++){
      b_ir0_stack->buf[j * N + i] = b_ir0->buf[j];
    }
  }
}

/*
 * Initialize the model.
 * Do input-independent job here.
 */
void namegen_initialize(int N, int rng_seed, char *parameter_fname) {

  /* Only the root process reads the parameter */ // 모든 파라메터를 가져온다.
  if (1) {
    size_t parameter_binary_size = 0;
    float *parameter =
        (float *)read_binary(parameter_fname, &parameter_binary_size);

    /* Network parameters */
    character_embedding =
        new Tensor({NUM_CHAR, EMBEDDING_DIM}, parameter + OFFSET0); // shape, buf의 시작위치인것같음.

    W_ir0 = new Tensor({HIDDEN_DIM, EMBEDDING_DIM}, parameter + OFFSET1); // shape and buff / 1024, 512
    W_iz0 = new Tensor({HIDDEN_DIM, EMBEDDING_DIM}, parameter + OFFSET2); // 1024, 512
    W_in0 = new Tensor({HIDDEN_DIM, EMBEDDING_DIM}, parameter + OFFSET3); // 1024, 512
    W_ir1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET4); // 1024, 1024
    W_iz1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET5); // 1024, 1024
    W_in1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET6); // 1024, 1024

    W_hr0 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET7);// 1024, 1024
    W_hz0 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET8);// 1024, 1024
    W_hn0 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET9);// 1024, 1024
    W_hr1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET10);// 1024, 1024
    W_hz1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET11);// 1024, 1024
    W_hn1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET12);// 1024, 1024

    b_ir0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET13);//1024
    b_iz0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET14);//1024
    b_in0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET15);//1024
    b_ir1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET16);//1024
    b_iz1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET17);//1024
    b_in1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET18);//1024

    b_hr0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET19);//1024
    b_hz0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET20);//1024
    b_hn0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET21);//1024
    b_hr1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET22);//1024
    b_hz1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET23);//1024
    b_hn1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET24);//1024

    b_ir0_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_iz0_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_in0_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_ir1_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_iz1_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_in1_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    stack_vector(BATCHSIZE, b_ir0_stack);
    stack_vector(BATCHSIZE, b_iz0_stack);
    stack_vector(BATCHSIZE, b_in0_stack);
    stack_vector(BATCHSIZE, b_ir1_stack);
    stack_vector(BATCHSIZE, b_iz1_stack);
    stack_vector(BATCHSIZE, b_in1_stack);

    b_hr0_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_hz0_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_hn0_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_hr1_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_hz1_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    b_hn1_stack = new Tensor({HIDDEN_DIM, BATCHSIZE});
    stack_vector(BATCHSIZE, b_hr0_stack);
    stack_vector(BATCHSIZE, b_hz0_stack);
    stack_vector(BATCHSIZE, b_hn0_stack);
    stack_vector(BATCHSIZE, b_hr1_stack);
    stack_vector(BATCHSIZE, b_hz1_stack);
    stack_vector(BATCHSIZE, b_hn1_stack);


    W_fc = new Tensor({NUM_CHAR, HIDDEN_DIM}, parameter + OFFSET25);//256,1024
    b_fc = new Tensor({NUM_CHAR}, parameter + OFFSET26);//256
    b_fc_stack = new Tensor({NUM_CHAR, BATCHSIZE});
    stack_vector(BATCHSIZE, b_fc_stack);

    /* input, activations, output, etc. */
    input = new Tensor({1, BATCHSIZE});//1
    emb_out = new Tensor({EMBEDDING_DIM, BATCHSIZE});//512

    hidden0 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    hidden1 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024

    r0 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    r1 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    z0 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    z1 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    n0 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    n1 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    f = new Tensor({NUM_CHAR, BATCHSIZE});//256

    rtmp00 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    rtmp01 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    rtmp02 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    rtmp03 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    rtmp04 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    rtmp10 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    rtmp11 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    rtmp12 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    rtmp13 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    rtmp14 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024

    ztmp00 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    ztmp01 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    ztmp02 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    ztmp03 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    ztmp04 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024
    ztmp10 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ztmp11 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ztmp12 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ztmp13 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ztmp14 = new Tensor({HIDDEN_DIM, BATCHSIZE});

    ntmp00 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp01 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp02 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp03 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp04 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp05 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp10 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp11 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp12 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp13 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp14 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    ntmp15 = new Tensor({HIDDEN_DIM, BATCHSIZE});

    htmp00 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    htmp01 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    htmp02 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    htmp10 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    htmp11 = new Tensor({HIDDEN_DIM, BATCHSIZE});
    htmp12 = new Tensor({HIDDEN_DIM, BATCHSIZE});//1024

    rfloats = new Tensor({N * MAX_LEN});// generate 글자 * 글자 수
    ftmp0 = new Tensor({NUM_CHAR, BATCHSIZE});//256
    char_prob = new Tensor({NUM_CHAR, BATCHSIZE});//256

    // int M_start = (N / mpi_size) * mpi_rank;
    // int M_end = (N / mpi_size) * (mpi_rank + 1);
    // if(mpi_rank == 0){
    //   #pragma omp parallel for num_threads(mpi_world_size - 1)
    //   for(int i = 1; i < mpi_size; i++)
    //   {
    //     // 1,2,3 rank의 start, end값은 모르기때문에 이렇게 처리해준다.
    //     int M_start = i * (N / mpi_size);
    //     int M_end = (i + 1) * (N / mpi_size);
    //     MPI_Send(A + M_start * K, (M_end - M_start) * K, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
    //     MPI_Send(B, K*N, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
    //   }

    // }else{
    //   MPI_Recv((void*)(A + M_start * K), (M_end - M_start) * K, MPI_FLOAT, 0, 1, MPI_COMM_WORLD,MPI_STATUS_IGNORE);
    //   MPI_Recv((void*)B, K*N, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    // }

    //cuda 준비


    //
    int size = N / mpi_size;
    if (mpi_rank == 0) {
      CUDA_CALL(hipGetDeviceCount(&num_devices));
      printf("size: %d\n",size);

      printf("Using %d devices\n", num_devices);
      for (int i = 0; i < num_devices; i++) {
        hipDeviceProp_t prop;
        CUDA_CALL(hipGetDeviceProperties(&prop, i));

        // Try printing more detailed information here
        printf("GPU %d: %s\n", i, prop.name);
      }

      if (num_devices <= 0) {
        printf("No CUDA device found. Aborting\n");
        exit(1);
      }

      // Setup problem size for each GPU
      for (int i = 0; i < num_devices; i++) {
        Mbegin[i] = (size / num_devices) * i;
        Mend[i] = Mbegin[i] + (size / num_devices);
        // printf("Mbegin[%d]: %d, Mend[%d]: %d\n",i,Mbegin[i],i,Mend[i]);
      }
      // Mend[num_devices - 1] = M;

      // Allocate device memory for each GPU
      for (int i = 0; i < num_devices; i++) {
        CUDA_CALL(hipSetDevice(i));
        CUDA_CALL(hipMalloc(&a_d[i], (Mend[i] - Mbegin[i]) * BATCHSIZE * sizeof(float))); // 이렇게 1024로 해도 되나?
        CUDA_CALL(hipMalloc(&b_d[i], BATCHSIZE * N * sizeof(float)));
        CUDA_CALL(hipMalloc(&c_d[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
      }
    }else{
      CUDA_CALL(hipGetDeviceCount(&num_devices));

      for (int i = 0; i < num_devices; i++) {
        hipDeviceProp_t prop;
        CUDA_CALL(hipGetDeviceProperties(&prop, i));
      }

      // Setup problem size for each GPU
      for (int i = 0; i < num_devices; i++) {
        Mbegin[i] = (size / num_devices) * i;
        Mend[i] = Mbegin[i] + (size / num_devices);
        // printf("Mbegin[%d]: %d, Mend[%d]: %d\n",i,Mbegin[i],i,Mend[i]);
      }

      for (int i = 0; i < num_devices; i++) {
        CUDA_CALL(hipSetDevice(i));
        CUDA_CALL(hipMalloc(&a_d[i], (Mend[i] - Mbegin[i]) * BATCHSIZE * sizeof(float)));
        CUDA_CALL(hipMalloc(&b_d[i], BATCHSIZE * N * sizeof(float)));
        CUDA_CALL(hipMalloc(&c_d[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
      }
    }
  }
}

/*
 * Generate names.
 * Any input-dependent computation/communication must be done here.
 * N: # of names to generate
 * random_floats: N*MAX_LEN sequence of random floats in [0,1].
 * output: 2D-array of size N x (MAX_LEN+1), allocaetd at main.cpp
 */
void namegen(int N, float *random_floats, char *output) { // 병렬화 대상

  // /* Only root process does the job, for now... */
  // if (mpi_rank != 0)
  //   return;
  int size = N / mpi_size;
  int M_start = (size) * mpi_rank;
  int M_end = (size) * (mpi_rank + 1);
// char_prob, rfloats, input, output, N, l)
  if (mpi_rank == 0){
    memcpy(rfloats->buf, random_floats, N * MAX_LEN * sizeof(float)); // buf = pointer to data, 메모리에 있는 random_floats를 buf로 옮긴다.
    memset(output, 0, N * (MAX_LEN + 1) * sizeof(char)); // 메모리 시작점부터 특정 범위까지 특정 값으로 지정할 수 있다.
    // input->set_SOS(); // start of sequence

    #pragma omp parallel for num_threads(mpi_size - 1)
    for(int i = 1; i < mpi_size; i++)
    {
      // int M_start = i * (N / mpi_size);
      // int M_end = (i + 1) * (N / mpi_size);
      MPI_Send(output + M_start* (MAX_LEN + 1), (M_end - M_start)* (MAX_LEN + 1), MPI_CHARACTER, i, 1, MPI_COMM_WORLD);
      MPI_Send(random_floats, N * MAX_LEN, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
    }
      /* Initialize input and hidden vector. */
    /* One hidden vector for each GRU layer */
    input->set_SOS();
    hidden0->set_zero(); // hidden0,1을 0으로 지정한다.
    hidden1->set_zero();

    for (int l = 0; l < MAX_LEN; l++) { // 이름의 최대 길이는 10이다.
      /* Embedding */
      embedding(input, character_embedding, emb_out); // 여기서 input은 buf[0]만을 가지고 있다. 만약 사이즈는 512다.

      // double namegen_st = get_time();
      // // /* First layer r */ // GRU연산을 보면 r, z, n을 구하고 이를 통해서 h를 구한다. 
      matmul(W_ir0, emb_out, rtmp00); // weight_input * embedding / NK * K -> N (matrix mector multiplication)
      matmul(W_hr0, hidden0, rtmp01); // weight_hidden * hidden / 
      elemwise_add(rtmp00, b_ir0_stack, rtmp02); // weight_input * embedding + bias_input / 
      elemwise_add(rtmp02, rtmp01, rtmp03); //weight_input * embedding + weight_hidden * hidden + bias_input
      elemwise_add(rtmp03, b_hr0_stack, rtmp04); // weight_input * embedding + weight_hidden * hidden + bias_input + bias_hidden  
      elemwise_sigmoid(rtmp04, r0); // activation(weight_input * embedding + weight_hidden * hidden + bias_input + bias_hidden) -> r0
      // double namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for first layer r: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* First layer z */
      matmul(W_iz0, emb_out, ztmp00);
      matmul(W_hz0, hidden0, ztmp01);
      elemwise_add(ztmp00, b_iz0_stack, ztmp02);
      elemwise_add(ztmp02, ztmp01, ztmp03);
      elemwise_add(ztmp03, b_hz0_stack, ztmp04);
      elemwise_sigmoid(ztmp04, z0);
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for first layer z: %.6f seconds\n", elapsed_time);
      // }
      
      // namegen_st = get_time();
      // /* First layer n */
      matmul(W_in0, emb_out, ntmp00);
      elemwise_add(ntmp00, b_in0_stack, ntmp01);
      matmul(W_hn0, hidden0, ntmp02);
      elemwise_add(ntmp02, b_hn0_stack, ntmp03);
      elemwise_mul(r0, ntmp03, ntmp04);
      elemwise_add(ntmp01, ntmp04, ntmp05);
      elemwise_tanh(ntmp05, n0);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", n0->shape[0], n0->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, n0->buf[j * N + i]);
      //   }
      // }      
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for first layer n: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* First layer h (hidden) */
      elemwise_oneminus(z0, htmp00); // 1 - z0
      elemwise_mul(htmp00, n0, htmp01); // vector * vector 
      elemwise_mul(z0, hidden0, htmp02);
      elemwise_add(htmp01, htmp02, hidden0);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", hidden0->shape[0], hidden0->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, hidden0->buf[j * N + i]);
      //   }
      // }     
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for first layer h: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Second layer r */ // GRU2
      matmul(W_ir1, hidden0, rtmp10);
      matmul(W_hr1, hidden1, rtmp11);
      elemwise_add(rtmp10, b_ir1_stack, rtmp12);
      elemwise_add(rtmp12, rtmp11, rtmp13);
      elemwise_add(rtmp13, b_hr1_stack, rtmp14);
      elemwise_sigmoid(rtmp14, r1);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", r1->shape[0], r1->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, r1->buf[j * N + i]);
      //   }
      // } 
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for second layer r: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Second layer z */
      matmul(W_iz1, hidden0, ztmp10);
      matmul(W_hz1, hidden1, ztmp11);
      elemwise_add(ztmp10, b_iz1_stack, ztmp12);
      elemwise_add(ztmp12, ztmp11, ztmp13);
      elemwise_add(ztmp13, b_hz1_stack, ztmp14);
      elemwise_sigmoid(ztmp14, z1);
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for second layer z: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Second layer n */
      matmul(W_in1, hidden0, ntmp10);
      elemwise_add(ntmp10, b_in1_stack, ntmp11);
      matmul(W_hn1, hidden1, ntmp12);
      elemwise_add(ntmp12, b_hn1_stack, ntmp13);
      elemwise_mul(r1, ntmp13, ntmp14);
      elemwise_add(ntmp11, ntmp14, ntmp15);
      elemwise_tanh(ntmp15, n1);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", n1->shape[0], n1->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, n1->buf[j * N + i]);
      //   }
      // }
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for second layer n: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Second layer h (hidden) */
      elemwise_oneminus(z1, htmp10);
      elemwise_mul(htmp10, n1, htmp11);
      elemwise_mul(z1, hidden1, htmp12);
      elemwise_add(htmp11, htmp12, hidden1);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", hidden1->shape[0], hidden1->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, hidden1->buf[j * N + i]);
      //   }
      // }
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for second layer h: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Fully connected layer */ // linear
      matmul(W_fc, hidden1, ftmp0);
      elemwise_add(ftmp0, b_fc_stack, f);

      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", f->shape[0], f->shape[1]);
      // //value
      // for(int i = 0; i < 1; i++){
      //   for(int j = 0; j < 1024; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, hidden1->buf[j * N + i]);
      //   }
      // }    
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for fully connected layer: %.6f seconds\n", elapsed_time);
      // }

    //   // namegen_st = get_time();
    //   // /* Softmax */
      softmax(f, char_prob, BATCHSIZE);
    //   // //shape
    //   // printf("\nshape of emb_out: <%zd, %zd>\n", char_prob->shape[0], char_prob->shape[1]);
    //   // //value
    //   // for(int i = 5; i < 8; i++){
    //   //   for(int j = 0; j < 10; j++){
    //   //     printf("%d번째 word %d번: %f\n",i, j+1, char_prob->buf[j * N + i]);
    //   //   }
    //   // } 
    // //   // namegen_en = get_time();
    // //   // if (mpi_rank == 0) {
    // //   //   double elapsed_time = namegen_en - namegen_st;
    // //   //   printf("Elapsed time for softmax: %.6f seconds\n", elapsed_time);
    // //   // }

    //   // namegen_st = get_time();
    //   // /* Random select */
      random_select(char_prob, rfloats, input, output, N, l); // 아 씨발
    //   // //shape
    //   // printf("\nshape of emb_out: <%zd, %zd>\n", char_prob->shape[0], char_prob->shape[1]);
    //   // //value
    //   // for(int i = 0; i < 1; i++){
    //   //   for(int j = 0; j < 256; j++){
    //   //     printf("%d번째 word %d번: %f\n",i, j+1, char_prob->buf[j * BATCHSIZE + i]);
    //   //   }
    //   // }
    //   // break;
    //   // namegen_en = get_time();
    //   // if (mpi_rank == 0) {
    //   //   double elapsed_time = namegen_en - namegen_st;
    //   //   printf("Elapsed time for random select: %.6f seconds\n", elapsed_time);
    //   // }
    }
    
    for(int i = 1; i < mpi_size; i++)
    {
      int M_start = i * (N / mpi_size);
      int M_end = (i + 1) * (N / mpi_size);
      // MPI_Recv(output + M_start * (MAX_LEN + 1),(M_end - M_start) * (MAX_LEN + 1), MPI_CHAR, i,i , MPI_COMM_WORLD, MPI_STATUS_IGNORE);
      printf("[mpi_rank:%d] recv address:%d\n",mpi_rank, M_start);
      printf("[mpi_rank:%d]amount of receieved message: %d\n", mpi_rank,(M_end - M_start));
      MPI_Recv((void*)(output + M_start * (MAX_LEN + 1)),  (M_end - M_start) * (MAX_LEN + 1), MPI_CHAR, i, i, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }
  }
  else if(mpi_rank != 0){
    char* output = (char *)malloc(N * (MAX_LEN + 1) * sizeof(char));
    float* random_floats_buff = (float *)malloc(N * (MAX_LEN) * sizeof(float));
    MPI_Recv((void*)(output + M_start * (MAX_LEN + 1)),  (M_end - M_start) * (MAX_LEN + 1), MPI_CHAR, 0, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    // MPI_Recv((void*)(random_floats_buff),  N * MAX_LEN, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    // MPI_Recv((void*)(input->buf + M_start),  (M_end - M_start), MPI_FLOAT, 0, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    MPI_Recv((void*)(random_floats_buff),  N * MAX_LEN, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    Tensor* rfloats = new Tensor({N * MAX_LEN});
    memcpy(rfloats->buf, random_floats_buff, N * MAX_LEN * sizeof(float));
    // memset(output, 0, N * (MAX_LEN + 1) * sizeof(char));
    /* Initialize input and hidden vector. */
    /* One hidden vector for each GRU layer */
    input->set_SOS();
    hidden0->set_zero(); // hidden0,1을 0으로 지정한다.
    hidden1->set_zero();

    for (int l = 0; l < MAX_LEN; l++) { // 이름의 최대 길이는 10이다.
      /* Embedding */
      embedding(input, character_embedding, emb_out); // 여기서 input은 buf[0]만을 가지고 있다. 만약 사이즈는 512다.

      // double namegen_st = get_time();
      // // /* First layer r */ // GRU연산을 보면 r, z, n을 구하고 이를 통해서 h를 구한다. 
      matmul(W_ir0, emb_out, rtmp00); // weight_input * embedding / NK * K -> N (matrix mector multiplication)
      matmul(W_hr0, hidden0, rtmp01); // weight_hidden * hidden / 
      elemwise_add(rtmp00, b_ir0_stack, rtmp02); // weight_input * embedding + bias_input / 
      elemwise_add(rtmp02, rtmp01, rtmp03); //weight_input * embedding + weight_hidden * hidden + bias_input
      elemwise_add(rtmp03, b_hr0_stack, rtmp04); // weight_input * embedding + weight_hidden * hidden + bias_input + bias_hidden  
      elemwise_sigmoid(rtmp04, r0); // activation(weight_input * embedding + weight_hidden * hidden + bias_input + bias_hidden) -> r0
      // double namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for first layer r: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* First layer z */
      matmul(W_iz0, emb_out, ztmp00);
      matmul(W_hz0, hidden0, ztmp01);
      elemwise_add(ztmp00, b_iz0_stack, ztmp02);
      elemwise_add(ztmp02, ztmp01, ztmp03);
      elemwise_add(ztmp03, b_hz0_stack, ztmp04);
      elemwise_sigmoid(ztmp04, z0);
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for first layer z: %.6f seconds\n", elapsed_time);
      // }
      
      // namegen_st = get_time();
      // /* First layer n */
      matmul(W_in0, emb_out, ntmp00);
      elemwise_add(ntmp00, b_in0_stack, ntmp01);
      matmul(W_hn0, hidden0, ntmp02);
      elemwise_add(ntmp02, b_hn0_stack, ntmp03);
      elemwise_mul(r0, ntmp03, ntmp04);
      elemwise_add(ntmp01, ntmp04, ntmp05);
      elemwise_tanh(ntmp05, n0);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", n0->shape[0], n0->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, n0->buf[j * N + i]);
      //   }
      // }      
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for first layer n: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* First layer h (hidden) */
      elemwise_oneminus(z0, htmp00); // 1 - z0
      elemwise_mul(htmp00, n0, htmp01); // vector * vector 
      elemwise_mul(z0, hidden0, htmp02);
      elemwise_add(htmp01, htmp02, hidden0);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", hidden0->shape[0], hidden0->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, hidden0->buf[j * N + i]);
      //   }
      // }     
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for first layer h: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Second layer r */ // GRU2
      matmul(W_ir1, hidden0, rtmp10);
      matmul(W_hr1, hidden1, rtmp11);
      elemwise_add(rtmp10, b_ir1_stack, rtmp12);
      elemwise_add(rtmp12, rtmp11, rtmp13);
      elemwise_add(rtmp13, b_hr1_stack, rtmp14);
      elemwise_sigmoid(rtmp14, r1);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", r1->shape[0], r1->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, r1->buf[j * N + i]);
      //   }
      // } 
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for second layer r: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Second layer z */
      matmul(W_iz1, hidden0, ztmp10);
      matmul(W_hz1, hidden1, ztmp11);
      elemwise_add(ztmp10, b_iz1_stack, ztmp12);
      elemwise_add(ztmp12, ztmp11, ztmp13);
      elemwise_add(ztmp13, b_hz1_stack, ztmp14);
      elemwise_sigmoid(ztmp14, z1);
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for second layer z: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Second layer n */
      matmul(W_in1, hidden0, ntmp10);
      elemwise_add(ntmp10, b_in1_stack, ntmp11);
      matmul(W_hn1, hidden1, ntmp12);
      elemwise_add(ntmp12, b_hn1_stack, ntmp13);
      elemwise_mul(r1, ntmp13, ntmp14);
      elemwise_add(ntmp11, ntmp14, ntmp15);
      elemwise_tanh(ntmp15, n1);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", n1->shape[0], n1->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, n1->buf[j * N + i]);
      //   }
      // }
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for second layer n: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Second layer h (hidden) */
      elemwise_oneminus(z1, htmp10);
      elemwise_mul(htmp10, n1, htmp11);
      elemwise_mul(z1, hidden1, htmp12);
      elemwise_add(htmp11, htmp12, hidden1);
      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", hidden1->shape[0], hidden1->shape[1]);
      // //value
      // for(int i = 5; i < 10; i++){
      //   for(int j = 0; j < 2; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, hidden1->buf[j * N + i]);
      //   }
      // }
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for second layer h: %.6f seconds\n", elapsed_time);
      // }

      // namegen_st = get_time();
      // /* Fully connected layer */ // linear
      matmul(W_fc, hidden1, ftmp0);
      elemwise_add(ftmp0, b_fc_stack, f);

      // //shape
      // printf("\nshape of emb_out: <%zd, %zd>\n", f->shape[0], f->shape[1]);
      // //value
      // for(int i = 0; i < 1; i++){
      //   for(int j = 0; j < 1024; j++){
      //     printf("%d번째 word %d번: %f\n",i, j+1, hidden1->buf[j * N + i]);
      //   }
      // }    
      // namegen_en = get_time();
      // if (mpi_rank == 0) {
      //   double elapsed_time = namegen_en - namegen_st;
      //   printf("Elapsed time for fully connected layer: %.6f seconds\n", elapsed_time);
      // }

    //   // namegen_st = get_time();
    //   // /* Softmax */
      softmax(f, char_prob, BATCHSIZE);
    //   // //shape
    //   // printf("\nshape of emb_out: <%zd, %zd>\n", char_prob->shape[0], char_prob->shape[1]);
    //   // //value
    //   // for(int i = 5; i < 8; i++){
    //   //   for(int j = 0; j < 10; j++){
    //   //     printf("%d번째 word %d번: %f\n",i, j+1, char_prob->buf[j * N + i]);
    //   //   }
    //   // } 
    // //   // namegen_en = get_time();
    // //   // if (mpi_rank == 0) {
    // //   //   double elapsed_time = namegen_en - namegen_st;
    // //   //   printf("Elapsed time for softmax: %.6f seconds\n", elapsed_time);
    // //   // }

    //   // namegen_st = get_time();
    //   // /* Random select */
      random_select(char_prob, rfloats, input, output, N, l); // 아 씨발
    //   // //shape
    //   // printf("\nshape of emb_out: <%zd, %zd>\n", char_prob->shape[0], char_prob->shape[1]);
    //   // //value
    //   // for(int i = 0; i < 1; i++){
    //   //   for(int j = 0; j < 256; j++){
    //   //     printf("%d번째 word %d번: %f\n",i, j+1, char_prob->buf[j * BATCHSIZE + i]);
    //   //   }
    //   // }
    //   // break;
    //   // namegen_en = get_time();
    //   // if (mpi_rank == 0) {
    //   //   double elapsed_time = namegen_en - namegen_st;
    //   //   printf("Elapsed time for random select: %.6f seconds\n", elapsed_time);
    //   // }
    }
    
    MPI_Send(output + M_start * (MAX_LEN + 1), (M_end - M_start) * (MAX_LEN + 1), MPI_CHAR, 0, mpi_rank, MPI_COMM_WORLD);
    printf("[mpi_rank:%d]amount of sended message: %d\n", mpi_rank,(M_end - M_start));
    printf("[mpi_rank:%d] send address:%d\n",mpi_rank, M_start);
  }

  // if(mpi_rank == 0){
  //   for(int i = 1; i < mpi_size; i++)
  //   {
  //     int M_start = i * (N / mpi_size);
  //     int M_end = (i + 1) * (N / mpi_size);
  //     // MPI_Recv(output + M_start * (MAX_LEN + 1),(M_end - M_start) * (MAX_LEN + 1), MPI_CHAR, i,i , MPI_COMM_WORLD, MPI_STATUS_IGNORE);
  //     printf("[mpi_rank:%d] recv address:%d\n",mpi_rank, M_start);
  //     printf("[mpi_rank:%d]amount of receieved message: %d\n", mpi_rank,(M_end - M_start));
  //     // MPI_Recv((void*)(output + M_start * (MAX_LEN + 1)),  (M_end - M_start) * (MAX_LEN + 1), MPI_CHAR, i, i, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
  //   }
  // }else{
  //   // MPI_Send(output + M_start * (MAX_LEN + 1), (M_end - M_start) * (MAX_LEN + 1), MPI_CHAR, 0, mpi_rank, MPI_COMM_WORLD);
  //   printf("[mpi_rank:%d]amount of sended message: %d\n", mpi_rank,(M_end - M_start));
  //   printf("[mpi_rank:%d] send address:%d\n",mpi_rank, M_start);
  // }
}

/*
 * Finalize the model.
 * Although it is not neccessary, we recommend to deallocate and destruct
 * everything you made in namegen_initalize() and namegen().
 */
void namegen_finalize() {
  if (1) {
    delete character_embedding;
    delete W_ir0;
    delete W_iz0;
    delete W_in0;
    delete W_ir1;
    delete W_iz1;
    delete W_in1;
    delete W_hr0;
    delete W_hz0;
    delete W_hn0;
    delete W_hr1;
    delete W_hz1;
    delete W_hn1;
    delete b_ir0;
    delete b_iz0;
    delete b_in0;
    delete b_ir1;
    delete b_iz1;
    delete b_in1;
    delete b_hr0;
    delete b_hz0;
    delete b_hn0;
    delete b_hr1;
    delete b_hz1;
    delete b_hn1;
    delete W_fc;
    delete b_fc;
    delete rfloats;

    delete input;
    delete emb_out;
    delete hidden0;
    delete hidden1;
    delete r0;
    delete r1;
    delete z0;
    delete z1;
    delete n0;
    delete n1;
    delete f;
    delete char_prob;
    delete rtmp00;
    delete rtmp01;
    delete rtmp02;
    delete rtmp03;
    delete rtmp04;
    delete rtmp10;
    delete rtmp11;
    delete rtmp12;
    delete rtmp13;
    delete rtmp14;
    delete ztmp00;
    delete ztmp01;
    delete ztmp02;
    delete ztmp03;
    delete ztmp04;
    delete ztmp10;
    delete ztmp11;
    delete ztmp12;
    delete ztmp13;
    delete ztmp14;
    delete ntmp00;
    delete ntmp01;
    delete ntmp02;
    delete ntmp03;
    delete ntmp04;
    delete ntmp05;
    delete ntmp10;
    delete ntmp11;
    delete ntmp12;
    delete ntmp13;
    delete ntmp14;
    delete ntmp15;
    delete htmp00;
    delete htmp01;
    delete htmp02;
    delete htmp10;
    delete htmp11;
    delete htmp12;
    delete ftmp0;
  }
}